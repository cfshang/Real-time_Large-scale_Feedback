
#include <iostream>
#include <stdio.h>
#include <hipfft/hipfft.h>

#include <cmath>

#include "EthFrmRcvPool.h"
#include <omp.h>
#include <pthread.h>
#include <sys/time.h>
#include <string.h>
#include "function.h"
using namespace std;
#define GPU_COMPUTE_CELL 2048*2048
#define PICTUREWIDTH 2048
#define PICTUREHEIGHT 2048

extern EthFrmRcvPool * pEthRcvPool[2];
//__device__ char data_dev_pointer[1024*1024*512];
//extern COMMON com[GPU_MAX];
extern TIF pretiff[GPU_MAX];
extern TIF tiff[GPU_MAX];


static int countReference[GPU_MAX] = {0,0};
void singleBuffDealReference(int number,size_t data_size,PARAMSTHREAD *para,unsigned int referenceNumber,unsigned int regNumber)
{
	//unsigned short *data_dev[GPU_MAX];


}
// use openMP
int prepareReference(int number,int len,PARAMSTHREAD *para,unsigned int _referenceNumber,unsigned int _regNumber)
{	

    //int count_gpu = 2;
    unsigned int referenceNumber= _referenceNumber;
    unsigned int regNumber= _regNumber;
    //referenceNumber
    //regNumber
	size_t data_size = len;
   // singleBuffDealReference(number,data_size,para,referenceNumber,regNumber);

    size_t size_dev[GPU_MAX];
    int indsBegin[GPU_MAX];
    hipEvent_t start[GPU_MAX],stop[GPU_MAX];
    float elapsedTime[GPU_MAX];
    //omp_set_num_threads(GPU_MAX);
//#pragma omp parallel for
    for(int i = 0; i < GPU_MAX; i++)
    {
        //std::cout << "copy to Device ok"  << "   ....   " << i << "...."<< endl;
        hipSetDevice(i);

        unsigned short *data_dev[2];
        unsigned short *data_host[2];
        short *data_index_dev[2];
        short *data_index_host[2];
        float *data_float_dev[2];
        float *data_float_host[2];
        //hipEventCreate(&start[i]);
        //hipEventCreate(&stop[i]);
        int width = para->com[i].width;
        int height = para->com[i].height;
        int length = para->com[i].length;
        //int length = width * height;
        tiff[i].width = width;
        tiff[i].height = height;
        tiff[i].length = length;
        int referenceOnePool = referenceNumber / 2;
        int tmp = referenceOnePool/ para->com[i].nFramePerRcvBuf;
        tmp = tmp * para->com[i].nFramePerRcvBuf;
        int num = para->com[i].nFramePerRcvBuf;
        if(number >= tmp )
            num = referenceOnePool-tmp;
        for(int j=0;j<2;j++){
            data_host[j] = pEthRcvPool[j]->curBufToProc_refer();
            data_index_host[j] = pEthRcvPool[j]->curBufIndexToProc_refer();
            data_float_host[j] = pEthRcvPool[j]->curBufFloatToProc_refer();
            hipError_t a = hipHostGetDevicePointer(&data_dev[j],data_host[j],0);
            if(a != hipSuccess){
                printf("The hipHostGetDevicePointer data_dev error in preparereference.cu, %d\n",i);
                exit(-1);
            }
            a = hipHostGetDevicePointer(&data_index_dev[j],data_index_host[j],0);
            if(a != hipSuccess){
                printf("The hipHostGetDevicePointer data_index_dev error in preparereference.cu, %d\n",j);
                exit(-1);
            }
            a = hipHostGetDevicePointer(&data_float_dev[j],data_float_host[j],0);
                if(a != hipSuccess){
                printf("The hipHostGetDevicePointer data_float_dev error in preparereference.cu, %d\n",j);
                exit(-1);
                }

            float *pfdata = thrust::raw_pointer_cast(&(para->com[i].fdata_d[0]));
            //thrust::device_vector<float> fdata_d(data_dev[j],data_dev[j]+data_size);
            //float *pfdata = thrust::raw_pointer_cast(&fdata_d[0]);
            hipMemcpyAsync(pfdata,data_float_host[j],data_size*sizeof(float),hipMemcpyHostToDevice,0);

            unsigned short *psdata = data_dev[j];
            short *dataIndex = data_index_dev[j];

            for(int k=0; k<num; k++){
                int refrIndex = dataIndex[k];
                PREREFERENCE *pre = &(para->pre_refr[i][refrIndex]);
                int index = k * length;
                tiff[i].fdata = pfdata + index;
                tiff[i].sdata = psdata + index;
                computeReference(tiff[i],pre,para->com[i],countReference[i]);
                countReference[i] ++;
            }
        }
        if(countReference[i] == referenceNumber){
            //thrust::device_vector<float> tmpNumXlat(tiff[i].length);
            float avenum = referenceNumber / regNumber;
            thrust::fill(para->com[i].tmpNumXlat.begin(),para->com[i].tmpNumXlat.end(),avenum);
            long int warpdataSize = para->com[i].warpNum* para->com[i].warpDataLen;
            //thrust::device_vector<float> tmpNumWarp(warpdataSize);
            thrust::fill(para->com[i].tmpNumWarp.begin(),para->com[i].tmpNumWarp.end(),avenum);
            //thrust::divides<float> opDivides;
            for(int k=0; k<regNumber; k++){
                thrust::transform(para->pre_refr[i][k].pre_xlat.begin(),para->pre_refr[i][k].pre_xlat.end(),para->com[i].tmpNumXlat.begin(),para->pre_refr[i][k].pre_xlat.begin(),complex_devides_functor());
                thrust::transform(para->pre_refr[i][k].pre_warp.begin(),para->pre_refr[i][k].pre_warp.end(),para->com[i].tmpNumWarp.begin(),para->pre_refr[i][k].pre_warp.begin(),complex_devides_functor());
            }
        }
    }

	return 1;
}
