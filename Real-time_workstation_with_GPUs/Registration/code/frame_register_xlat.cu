#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <vector>
#include "function.h"
#include <assert.h>
using namespace std;
#define BLOCK_NUM_SHIFT 512
#define THREAD_NUM_SHIFT 1024

__global__ void computeSum(float *out, const float *in, size_t N)
{
	// lenght = threads (BlockDim.x)
	__shared__ float sPartials[1024];
	// __shared__ int sindex[1024];
	float sum = 0;
	// int maxIndex = 0;
	const int tid = threadIdx.x;

	for (size_t i = blockIdx.x * blockDim.x + tid; i < N; i += blockDim.x * gridDim.x)
	{
		sum += in[i];
	}
	sPartials[tid] = sum;
	__syncthreads();

	for (int activeTrheads = blockDim.x / 2; activeTrheads > 0; activeTrheads /= 2)
	{
		if (tid < activeTrheads)
		{
			sPartials[tid] += sPartials[tid + activeTrheads];
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		out[blockIdx.x] = sPartials[0];
	}
}
__global__ void computeMean(float *out, const float *in,float *sum, size_t N)
{
	float ave = sum[0] / N;
	const int tid = threadIdx.x;

	for (size_t i = blockIdx.x * blockDim.x + tid; i < N; i += blockDim.x * gridDim.x)
	{
		out[i] = in[i] - ave;
	}
}
__global__ void cufftshift(float *src,float *out,int width,int height){
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	size_t xshift = width / 2;
	size_t yshift = height / 2;
	size_t size = width * height;
	int threadIndex = bid *BLOCK_NUM_SHIFT + tid;
	int offset = BLOCK_NUM_SHIFT*THREAD_NUM_SHIFT;
	for(int i=threadIndex; i< size; i+= offset){
		size_t yIdx = i / width;
		size_t xIdx = i % width;
		size_t outY = (yIdx + yshift) % height;
		size_t outX = (xIdx + xshift) % width;
		size_t outIdx = outX + width*outY;
		out[outIdx] = src[i];
	}

}
__global__ void findMaxIndex(float *out, const float *in, size_t N,int *index)
{
	// lenght = threads (BlockDim.x)
	__shared__ float sPartials[1024];
	__shared__ int sindex[1024];
	float max = in[0];
	int maxIndex = 0;
	const int tid = threadIdx.x;

	for (size_t i = blockIdx.x * blockDim.x + tid; i < N; i += blockDim.x * gridDim.x)
	{
		if(in[i] > max){
			max = in[i];
			maxIndex = i;
		}
	}
	sPartials[tid] = max;
	sindex[tid] = maxIndex;
	__syncthreads();

	for (int activeTrheads = blockDim.x / 2; activeTrheads > 0; activeTrheads /= 2)
	{
		if (tid < activeTrheads)
		{
			if( sPartials[tid] < sPartials[tid + activeTrheads]){
				sPartials[tid] = sPartials[tid + activeTrheads];
				sindex[tid] = sindex[tid + activeTrheads];
			}
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		out[blockIdx.x] = sPartials[0];
		index[blockIdx.x]  = sindex[0];
	}
}
__global__ void imageShift(float *fdest,unsigned short *sdest,int width,int height,int *partial_index,int *maxindex){
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	int index = partial_index[maxindex[0]];
	int maxx = index % width;
	int maxy = index / width;
	int tmp = ((double)width) / 2- 0.5 + 1;
	int dx = maxx +1 - tmp - 1;
	tmp = (double(height)) / 2- 0.5 + 1;
	int dy = maxy +1 - tmp - 1;
	size_t size = width * height;
	int threadIndex = bid *BLOCK_NUM_SHIFT + tid;
	int offset = BLOCK_NUM_SHIFT*THREAD_NUM_SHIFT;

	if(dy >= 0 && dx >= 0){
		for(int i=threadIndex; i< size; i+= offset){
			int desty = i / width;
			int destx = i % width;
			if(desty < (height-dy) && destx < (width-dx)){
				int srcy = desty + dy;
				int srcx = destx + dx;
				int src = srcy * width + srcx;
				// fdest[i] = (float)ssrc[src];
				sdest[i] = (unsigned short)fdest[src];
			}
		}
	}
	else if(dy >= 0 && dx <=0){
		for(int i=threadIndex; i< size; i+= offset){
			int desty = i / width;
			int destx = i % width;
			if(desty < (height-dy) && destx >= -dx){
				int srcy = desty + dy;
				int srcx = destx + dx;
				int src = srcy * width + srcx;
				// fdest[i] = (float)ssrc[src];
				sdest[i] = (unsigned short)fdest[src];
			}
		}

	}
	else if(dy <= 0 && dx >=0){
		for(int i=threadIndex; i< size; i+= offset){
			int desty = i / width;
			int destx = i % width;
			if(desty >= -dy && destx <(width - dx)){
				int srcy = desty + dy;
				int srcx = destx + dx;
				int src = srcy * width + srcx;
				// fdest[i] = (float)ssrc[src];
				sdest[i] = (unsigned short)fdest[src];
			}
		}

	}
	else if(dy <= 0 && dx <=0){

		for(int i=threadIndex; i< size; i+= offset){
			int desty = i / width;
			int destx = i % width;
			if(desty >= -dy && destx >= -dx){
				int srcy = desty + dy;
				int srcx = destx + dx;
				int src = srcy * width + srcx;
				// fdest[i] = (float)ssrc[src];
				sdest[i] = (unsigned short)fdest[src];
			}
		} 
	}


}
__global__ void imageShift_fdata(float *fdest,unsigned short *sdata,int width,int height){
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	int threadIndex = bid *blockDim.x + tid;
	int offset = blockDim.x * gridDim.x;
	int size = width * height;
	for(int i=threadIndex; i< size; i+= offset){
		fdest[i] = (float)sdata[i];
	}
}
void frameRegisterXlat(PREREFERENCE *pre_refr,TIF *tiff,COMMON &com){
	struct timeval start,stop;
	float diff;
	// gettimeofday(&start,NULL);
	int width = tiff->width;
	int height = tiff->height;

	float *fdata = tiff->fdata;
	int resSize = tiff->length;

	//thrust::device_vector<float> meandata(tiff->length);
	// float *raw_point_data = com.raw_point_data;
	// thrust::inclusive_scan(ddata.begin(),ddata.end(),sumdata.begin());
	// thrust::device_vector<float> partial_sum_d(4096);
	//   float *partial_sum = com.partial_sum;
	computeSum<<<4096,1024>>>(com.partial_sum,fdata,resSize);
	// thrust::device_vector<float> sum_d(1);


	computeSum<<<1,1024>>>(com.sum,com.partial_sum,4096);
	//float sum_data = com.sum[0];
	computeMean<<<BLOCK_NUM_SHIFT,THREAD_NUM_SHIFT>>>(com.raw_point_data,fdata,com.sum,resSize);


	//thrust::device_vector<Complex> d_res_fft(tiff->length);
	//Complex *raw_point_res_fft = thrust::raw_pointer_cast(&d_res_fft[0]);

	//fft(raw_point_data,raw_point_res_fft,tiff->width,tiff->height);
	hipfftExecR2C(com.plan_big,com.raw_point_data,com.raw_point_res_fft);

	//thrust::device_vector<Complex> pre_xlat_fft(pre_refr->pre_xlat,pre_refr->pre_xlat+resSize);
	//Complex *raw_pre_xlat_fft = thrust::raw_pointer_cast(&pre_refr->pre_xlat[0]);
	//thrust::device_vector<Complex> resDot(resSize);
	//Complex *raw_resDot = thrust::raw_pointer_cast(&resDot[0]);
	thrust::transform(pre_refr->pre_xlat.begin(),pre_refr->pre_xlat.end(),com.d_res_fft.begin(),com.resDot.begin(),complex_multiplies_functor());

	//thrust::device_vector<float> ifft_res(resSize);
	//float *raw_ifft_res = thrust::raw_pointer_cast(&ifft_res[0]);
	//ifft(raw_resDot,raw_ifft_res,width,height);
	hipfftExecC2R(com.iplan_big,com.raw_resDot,com.raw_ifft_res);

	//thrust::device_vector<float> fftshift_res(resSize);
	//float *raw_fftshift_res = thrust::raw_pointer_cast(&fftshift_res[0]);
	//fftshift2D(h_raw_ifft_res,width,height);
	cufftshift<<<BLOCK_NUM_SHIFT,THREAD_NUM_SHIFT,0>>>(com.raw_ifft_res,com.raw_fftshift_res,tiff->width,tiff->height);
	//thrust::device_vector<float> partial_d(4096);
	// *partial = thrust::raw_pointer_cast(&partial_d[0]);
	//thrust::device_vector<int> partial_index_d(4096);
	//int *partial_index = thrust::raw_pointer_cast(&partial_index_d[0]);
	//unsigned int sharedSize = 1024 *sizeof(float);
	findMaxIndex<<<4096,1024>>>(com.partial_sum,com.raw_fftshift_res,resSize,com.partial_index);

	// cout << max_h[589] << endl;
    //thrust::device_vector<float> max_d(1);
    //float *max = thrust::raw_pointer_cast(&max_d[0]);
    //thrust::device_vector<int> maxindex_d(1);
    //int *maxindex = thrust::raw_pointer_cast(&maxindex_d[0]);
    findMaxIndex<<<1,1024>>>(com.maxdata,com.partial_sum,4096,com.maxindex);
	//MAXINDEX maxIndex;

	//findMaxIndex(h_raw_ifft_res,height,width,maxIndex);

    //thrust::device_vector<unsigned short> sdata(tiff->sdata,tiff->sdata+tiff->length);
    //unsigned short *ssrc = thrust::raw_pointer_cast(&sdata[0]);

    imageShift<<<1024,THREAD_NUM_SHIFT,0>>>(tiff->fdata,tiff->sdata,tiff->width,tiff->height,com.partial_index,com.maxindex);
    imageShift_fdata<<<1024,THREAD_NUM_SHIFT,0>>>(tiff->fdata,tiff->sdata,tiff->width,tiff->height);
    //gettimeofday(&stop,NULL);
    //diff = (stop.tv_sec-start.tv_sec)*1000000+(stop.tv_usec-start.tv_usec);
   // cout << " ********************** the whole image cost : "  << diff << " us. "<< endl;
	if(DEBUG)
		cout << "frame registration xlat ok!" << endl;
	
}
