#include "hip/hip_runtime.h"
#include "function.h"

using namespace std;

#define BLOCK_NUM_ISHIFT 256
#define THREAD_NUM_ISHIFT 256
#define OPENMP_THREAD_NUM 12
__global__ void getwarpdata_kernel(float *data,float *warpdata,int width,int *index,int warpSize,int warpDataLen,int warpNum){
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	// int length = width * height;
	//   int threadIndex = bid *BLOCK_NUM_SHIFT + tid;
	int offset = blockDim.x*gridDim.x;
	/*int srcstart = index[bid];
	  int deststart = bid * warpDataLen;

	  for(int i=0; i< warpSize;i++){
	  int destIndex = deststart+tid*warpSize+i;
	  int srcIndex  = srcstart+tid*width+i;
	  warpdata[destIndex] = data[srcIndex];
	  }*/
	int srcstart;
	int deststart;
	int destindex;
	int srcindex;
	for(int i=0;i <warpNum;i++){
		srcstart = index[i];
		deststart = i * warpDataLen;
		destindex = deststart + bid * warpSize + tid;
		srcindex = srcstart + bid *width + tid;
		warpdata[destindex] = data[srcindex];
	}
}
__global__ void cuifftshift(float *src,float *out,int width,int height){
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	//size_t size = width * height;
	size_t xshift = width / 2;
	if(width % 2 != 0)
		xshift ++ ;
	size_t yshift = height / 2;
	if(height % 2 != 0)
		yshift ++;
	size_t size = width * height;
	int threadIndex = bid *blockDim.x + tid;
	int offset = blockDim.x*gridDim.x;
	for(int i=threadIndex; i< size; i+= offset){
		size_t yIdx = i / width;
		size_t xIdx = i % width;
		size_t outY = (yIdx + yshift) % height;
		size_t outX = (xIdx + xshift) % width;
		size_t outIdx = outX + width*outY;
		out[outIdx] = src[i];
	}

}
__global__ void cuifftshift1(float *src,float *out,int width,int height,int warpNum, int warpdataLen){
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	//size_t size = width * height;
	size_t xshift = width / 2;
	if(width % 2 != 0)
		xshift ++ ;
	size_t yshift = height / 2;
	if(height % 2 != 0)
		yshift ++;
	size_t size = width * height;
	int threadIndex = bid *blockDim.x + tid;
	int offset = blockDim.x*gridDim.x;
	for(int j = 0; j < warpNum; j ++){
		int index = j * warpdataLen;
		for(int i=threadIndex; i< size; i+= offset){
			size_t yIdx = i / width;
			size_t xIdx = i % width;
			size_t outY = (yIdx + yshift) % height;
			size_t outX = (xIdx + xshift) % width;
			size_t outIdx = outX + width*outY;
			out[outIdx] = src[i];
		}
		__threadfence();
		__syncthreads();
	}

}
__global__ void cuComplexMulti(Complex *src1,Complex *src2,Complex *out,int warpNum,int warpdatalen){
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	int threadIndex = bid *blockDim.x + tid;
	int offset = blockDim.x*gridDim.x;
	int size = warpNum * warpdatalen;
	for(int i=threadIndex; i< size; i+= offset){
		out[i].x=(src1[i].x*src2[i].x - src1[i].y*src2[i].y);
		out[i].y=(src1[i].x*src2[i].y + src1[i].y*src2[i].x);
	}
	/*int index = bid * width + tid;
	  out[index].x=(src1[index].x*src2[index].x - src1[index].y*src2[index].y);
	  out[index].y=(src1[index].x*src2[index].y + src1[index].y*src2[index].x);*/
}
__global__ void cuComplexConj(Complex *src,Complex *out,int size){
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	int threadIndex = bid *blockDim.x + tid;
	int offset = blockDim.x*gridDim.x;
	for(int i=threadIndex; i< size; i+= offset){
		out[i].x=src[i].x;
		out[i].y= -src[i].y;
	}
}
__global__ void findWarpMaxIndex(float *out, const float *in, size_t N,int *index)
{
	// lenght = threads (BlockDim.x)
	__shared__ float sPartials[1024];
	__shared__ int sindex[1024];
	float max = in[0];
	int maxIndex = 0;
	const int tid = threadIdx.x;

	for (size_t i = blockIdx.x * blockDim.x + tid; i < N; i += blockDim.x * gridDim.x)
	{
		if(in[i] > max){
			max = in[i];
			maxIndex = i;
		}
	}
	sPartials[tid] = max;
	sindex[tid] = maxIndex;
	__syncthreads();

	for (int activeTrheads = blockDim.x / 2; activeTrheads > 0; activeTrheads /= 2)
	{
		if (tid < activeTrheads)
		{
			if( sPartials[tid] < sPartials[tid + activeTrheads]){
				sPartials[tid] = sPartials[tid + activeTrheads];
				sindex[tid] = sindex[tid + activeTrheads];
			}
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		out[blockIdx.x] = sPartials[0];
		index[blockIdx.x]  = sindex[0];
	}
}
__global__ void cuComputeDxDy(int inIndex,int *maxIndex,int *dx,int *dy,int warpSize,int squareSize,int outIndex,int maximumShift){
	const int tid = threadIdx.x;
	// const int bid = blockIdx.x;
	//  int threadIndex = bid *blockDim.x + tid;
	//int offset = blockDim.x*gridDim.x;
	// int size = width * height;
	int x = maxIndex[inIndex] % warpSize;
	int y = maxIndex[inIndex] / warpSize;
	dx[outIndex] = x+1 - squareSize - 2;
	dy[outIndex] = y+1 - squareSize - 2;
	if(abs(dx[outIndex]) > maximumShift)
		dx[outIndex] = 0;
	if(abs(dy[outIndex]) > maximumShift)
		dy[outIndex] = 0;
}
__global__ void cuComputeDxDy1(int *maxIndex,int *dx,int *dy,int warpSize,int squareSize,int maximumShift){
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	int threadIndex = bid *blockDim.x + tid;
	//int offset = blockDim.x*gridDim.x;
	// int size = width * height;
	int x = maxIndex[threadIndex] % warpSize +1;
	int y = maxIndex[threadIndex] / warpSize +1;
	dx[threadIndex] = x+1 - squareSize - 2;
	dy[threadIndex] = y+1 - squareSize - 2;
	if(abs(dx[threadIndex]) > maximumShift)
		dx[threadIndex] = 0;
	if(abs(dy[threadIndex]) > maximumShift)
		dy[threadIndex] = 0;
}
__global__ void cuComputeDxDy2(int *maxIndex,int *dx,int *dy,int warpSize,int squareSize,int maximumShift,int warpNum){
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	int threadIndex = bid *blockDim.x + tid;
	//int offset = blockDim.x*gridDim.x;
	int size = warpNum;
	int offset = blockDim.x*gridDim.x;
	for(int i=threadIndex; i< size; i+= offset){
		int x = maxIndex[i] % warpSize +1;
		int y = maxIndex[i] / warpSize +1;
		dx[i] = x+1 - squareSize - 2;
		dy[i] = y+1 - squareSize - 2;
		if(abs(dx[i]) > maximumShift)
			dx[i] = 0;
		if(abs(dy[i]) > maximumShift)
			dy[i] = 0;
	}
}
__global__ void getmaxindexarray(int *out,int *src,int *srcindex,int index){
	out[index] = src[srcindex[0]];
}
void frameRegisterWarp(PREREFERENCE *pre_refr,TIF *tiff,COMMON &com){
	unsigned int width = tiff->width;
	unsigned int height = tiff->height;
	int length = tiff->length;
	//long int  warpdataSize = com.warpNum*com.warpDataLen;
	struct timeval start,stop;
	float diff;


	//20ms	
	//   gettimeofday(&start,NULL);
	// getwarpdata_kernel<<<com.warpNum,com.warpSize>>>(tiff->fdata,tiff->warpdata,com.para_d[2] ,com.raw_beginIndex_d,com.para_d[1],com.para_d[0]);
	getwarpdata_kernel<<<com.warpSize,com.warpSize>>>(tiff->fdata,tiff->warpdata,com.para_d[2] ,com.raw_beginIndex_d,com.para_d[1],com.para_d[0],com.para_d[6]);
	hipDeviceSynchronize();
	//   gettimeofday(&stop,NULL);
	//   diff = (stop.tv_sec-start.tv_sec)*1000+(stop.tv_usec-start.tv_usec)/1000;
	//   cout << "getwarpdata cost " << diff << " ms "<< endl;


	//hipEvent_t startd,stopd;
	//float elapsedTimed;
	//hipEventCreate(&startd);
	//hipEventCreate(&stopd);
	//   gettimeofday(&start,NULL);
	//hipEventRecord(startd,0);

	/*    for(int i=0;i<com.warpNum;i++){
	//int count = i * com.warpNumHeight + j;
	int count = i;
	int index = count * com.warpDataLen;
	float *point_in = tiff->warpdata+index;
	Complex *point_out = com.warp+index;

	hipfftExecR2C(com.plan_small,point_in,point_out);



	}*/
	//

	hipfftExecR2C(com.plan_batch,tiff->warpdata,com.warp);
	hipDeviceSynchronize();
	//   gettimeofday(&stop,NULL);
	//   diff = (stop.tv_sec-start.tv_sec)*1000000+(stop.tv_usec-start.tv_usec);
	//   cout << "fft  cost " << diff << " us "<< endl;
	//}
	//hipfftExecR2C(com.plan_batch,tiff->warpdata,warp);
	//hipEventRecord(stopd,0);
	//hipEventSynchronize(stopd);
	//hipEventElapsedTime(&elapsedTimed,startd,stopd);
	//cout << "fft times  " << " : " << elapsedTimed << "ms." << endl;

	//   gettimeofday(&start,NULL);
	thrust::transform(com.d_warp.begin(),com.d_warp.end(),com.d_warp_conj.begin(),device_conj_functor());
	// cuComplexConj<<<256,256>>>(com.warp,com.warp_conj,com.warpdatasize_d[0]);
	//   gettimeofday(&stop,NULL);
	//   diff = (stop.tv_sec-start.tv_sec)*1000+(stop.tv_usec-start.tv_usec)/1000;
	//   cout << "conj cost " << diff << " ms "<< endl;


	//    gettimeofday(&start,NULL);
	Complex *prewarp = thrust::raw_pointer_cast(&pre_refr->pre_warp[0]);

	cuComplexMulti<<<1024,1024>>>(com.warp_conj,prewarp,com.warpData,com.warpNum,com.warpDataLen);

	// gettimeofday(&start,NULL);

	hipfftExecC2R(com.iplan_batch,com.warpData,com.raw_ifftres_warpdata);
	hipDeviceSynchronize();
	// gettimeofday(&stop,NULL);
	// diff = (stop.tv_sec-start.tv_sec)*1000000+(stop.tv_usec-start.tv_usec);
	// cout << "ifft  cost " << diff << " us "<< endl;

	cuifftshift1<<<256,256>>>(com.raw_ifftres_warpdata,com.raw_ishift_warpdata,com.para_d[1],com.para_d[1],com.para_d[6],com.para_d[0]);

	thrust::device_vector<int> dxArray(com.warpNum);
	thrust::device_vector<int> dyArray(com.warpNum);
	int *dx = thrust::raw_pointer_cast(&dxArray[0]);
	int *dy = thrust::raw_pointer_cast(&dyArray[0]);
	thrust::device_vector<int> mInd(com.warpNum);
	int *mindex = thrust::raw_pointer_cast(&mInd[0]);
	// #pragma omp parallel for
	//  thrust::device_vector<Complex> d_dotRes(com.warpDataLen);
	//  Complex *warpData = thrust::raw_pointer_cast(&d_dotRes[0]);
	//   thrust::device_vector<float> C(com.warpDataLen);
	//  float *raw_c = thrust::raw_pointer_cast(&C[0]);
	//  thrust::device_vector<float> d_c(com.warpDataLen);
	//  float *d_raw_c = thrust::raw_pointer_cast(&d_c[0]);
	thrust::device_vector<float> partial_d(256);
	float *partial_small = thrust::raw_pointer_cast(&partial_d[0]);
	thrust::device_vector<int> partial_index_d(256);
	int *partial_index_small = thrust::raw_pointer_cast(&partial_index_d[0]);
	unsigned int sharedSize = 256 *sizeof(float);
	thrust::device_vector<float> max_d(1);
	float *max_small = thrust::raw_pointer_cast(&max_d[0]);
	thrust::device_vector<int> maxindex_d(1);
	int *maxindex_small = thrust::raw_pointer_cast(&maxindex_d[0]);
	// hipfftResult r;

	//   gettimeofday(&stop,NULL);
	//   diff = (stop.tv_sec-start.tv_sec)*1000000+(stop.tv_usec-start.tv_usec);
	//   cout << "variable declarations: " << diff << " us "<< endl;

	//    gettimeofday(&start,NULL);
	for(int i=0; i< com.warpNum; i++){
		int index = i * com.warpDataLen;
		//r = hipfftXtExec(com.plan_xt_r2c,)
		//  Complex *src1 = com.warp_conj+index;
		//	Complex *src2 = prewarp+index;
		//       Complex *warpData = com.warpData + index;
		// gettimeofday(&start,NULL);
		//      cuComplexMulti<<<256,256,0>>>(src1,src2,warpData,com.para_d[1],com.para_d[1]);
		// gettimeofday(&stop,NULL);
		// diff = (stop.tv_sec-start.tv_sec)*1000000+(stop.tv_usec-start.tv_usec);
		// cout << "********************************************* multi  cost " << diff << " us "<< endl;
		// thrust::transform(d_wap.begin(),d_wap.end(),d_pre_warp.begin(),d_dotRes.begin(),complex_multiplies_functor());

		//Complex *warpData = thrust::raw_pointer_cast(&d_dotRes[0]);
		//  gettimeofday(&start,NULL);
		//        hipfftExecC2R(com.iplan_small,warpData,raw_c);
		//  gettimeofday(&stop,NULL);
		//  diff = (stop.tv_sec-start.tv_sec)*1000000+(stop.tv_usec-start.tv_usec);
		// cout << " *********************************************ifft cost " << diff << " us "<< endl;

		//   gettimeofday(&start,NULL);
		//     float *raw_c = com.raw_ifftres_warpdata + index;
		//     cuifftshift<<<com.warpSize,com.warpSize,0>>>(raw_c,d_raw_c,com.para_d[1],com.para_d[1]);
		//  gettimeofday(&stop,NULL);
		//  diff = (stop.tv_sec-start.tv_sec)*1000000+(stop.tv_usec-start.tv_usec);
		//  cout << "*********************************************cusiffthift  cost " << diff << " us "<< endl;

		// gettimeofday(&start,NULL);
		float *d_raw_c = com.raw_ishift_warpdata + index;
		findWarpMaxIndex<<<256,256>>>(partial_small,d_raw_c,com.para_d[0],partial_index_small);

		findWarpMaxIndex<<<1,256>>>(max_small,partial_small,256,maxindex_small);
		// cuComputeDxDy<<<1,1>>>(maxindex_d[0],partial_index,dx,dy,para_d[1],para_d[4],i,para_d[5]);
		// mInd[i] = partial_index[maxindex_d[0]];
		// gettimeofday(&stop,NULL);
		//  diff = (stop.tv_sec-start.tv_sec)*1000000+(stop.tv_usec-start.tv_usec);
		//  cout << "*********************************************findmax cost " << diff << " us "<< endl;

		//  gettimeofday(&start,NULL);
		getmaxindexarray<<<1,1>>>(mindex,partial_index_small,maxindex_small,i);
		//  gettimeofday(&stop,NULL);
		//  diff = (stop.tv_sec-start.tv_sec)*1000000+(stop.tv_usec-start.tv_usec);
		//  cout << "*********************************************manage maxindex cost " << diff << " us "<< endl;
	}
//   gettimeofday(&stop,NULL);
//   diff = (stop.tv_sec-start.tv_sec)*1000+(stop.tv_usec-start.tv_usec)/1000;
//   cout << "register cost " << diff << " ms "<< endl;


//int *mindex = thrust::raw_pointer_cast(&mInd[0]);
//  gettimeofday(&start,NULL);
cuComputeDxDy1<<<1,com.warpNum,0>>>(mindex,dx,dy,com.para_d[1],com.para_d[4],com.para_d[5]);
//    cuComputeDxDy2<<<1,512>>>(mindex,dx,dy,com.para_d[1],com.para_d[4],com.para_d[5],com.para_d[6]);

//gettimeofday(&stop,NULL);
// diff = (stop.tv_sec-start.tv_sec)*1000000+(stop.tv_usec-start.tv_usec);
// cout << " *********** the small fft cost " << diff << " us "  << com.warpNum << endl;

//   gettimeofday(&start,NULL);
frameReformatWarpdata(tiff,dx,dy,com);
//   gettimeofday(&stop,NULL);
//   diff = (stop.tv_sec-start.tv_sec)*1000+(stop.tv_usec-start.tv_usec)/1000;
//   cout << " *********** ************** the small shift cost " << diff << " ms "  << com.warpNum << endl;

}
