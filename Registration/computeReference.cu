#include "hip/hip_runtime.h"

#include "function.h"

using namespace std;
__global__ void getwarpdata_k(float *data,float *warpdata,int width,int *index,int warpSize,int warpDataLen){
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	// int length = width * height;
	//   int threadIndex = bid *BLOCK_NUM_SHIFT + tid;
	int srcstart = index[bid];
	int deststart = bid * warpDataLen;
	for(int i=0; i< warpSize;i++){
		int destIndex = deststart+tid*warpSize+i;
		int srcIndex  = srcstart+tid*width+i;
		warpdata[destIndex] = data[srcIndex];
	}
	// int offset = BLOCK_NUM_SHIFT*THREAD_NUM_SHIFT;
	//  for(int i=threadIndex; i< size; i+= offset){
}

void computeReference(TIF &tiff,PREREFERENCE *pre_refr,COMMON &com,int count){

    //float *fdata;
	hipError_t a;

    //fdata = tiff.fdata;

    /*thrust::minus<float> opMinus;

	thrust::device_vector<float> ddata(fdata,fdata+tiff.length);
	thrust::device_vector<float> sumdata(tiff.length);
	thrust::device_vector<float> meandata(tiff.length);
	thrust::inclusive_scan(ddata.begin(),ddata.end(),sumdata.begin());

	float ave = sumdata[tiff.length-1] / tiff.length;

	thrust::device_vector<float> avedata(tiff.length);
	thrust::fill(avedata.begin(),avedata.end(),ave);

	thrust::transform(ddata.begin(),ddata.end(),avedata.begin(),meandata.begin(),opMinus);

	float *raw_point_data = thrust::raw_pointer_cast(&meandata[0]);

	unsigned int resSize = tiff.length;

    thrust::device_vector<Complex> d_res_fft(tiff.length);
    Complex *raw_point_res_fft = thrust::raw_pointer_cast(&d_res_fft[0]);*/
	// fftinit(tiff.width,tiff.height);

    int width = tiff.width;
    int height = tiff.height;

    float *fdata = tiff.fdata;
    int resSize = tiff.length;

    //thrust::device_vector<float> meandata(tiff->length);
    // float *raw_point_data = com.raw_point_data;
    // thrust::inclusive_scan(ddata.begin(),ddata.end(),sumdata.begin());
    // thrust::device_vector<float> partial_sum_d(4096);
    //   float *partial_sum = com.partial_sum;
    computeSum<<<4096,1024>>>(com.partial_sum,fdata,resSize);
    // thrust::device_vector<float> sum_d(1);


    computeSum<<<1,1024>>>(com.sum,com.partial_sum,4096);
    //float sum_data = com.sum[0];
    computeMean<<<BLOCK_NUM_SHIFT,THREAD_NUM_SHIFT>>>(com.raw_point_data,fdata,com.sum,resSize);
//	fft(raw_point_data,raw_point_res_fft,tiff.width,tiff.height);
   // hipfftExecR2C(com.plan_big,raw_point_data,raw_point_res_fft);
    hipfftExecR2C(com.plan_big,com.raw_point_data,com.raw_point_res_fft);

	//thrust::device_vector<Complex> d_res_conj(tiff.length);
	//if(count == 0)
	//	pre_refr->pre_xlat.resize(tiff.length);

    //thrust::device_vector<Complex> d_xlat_tmp(tiff.length);
    //Complex *raw_point_res_conj = thrust::raw_pointer_cast(&d_xlat_tmp[0]);

    thrust::transform(com.d_res_fft.begin(),com.d_res_fft.end(),com.d_xlat_tmp.begin(),device_conj_functor());
	//cout <<"-----------------------------" << pre_refr->pre_xlat.size() << endl;
    thrust::transform(com.d_xlat_tmp.begin(),com.d_xlat_tmp.end(),pre_refr->pre_xlat.begin(),pre_refr->pre_xlat.begin(),complex_plus_functor());
	pre_refr->pre_xlat_size = resSize;
	pre_refr->pre_xlat_xsize = tiff.width;
	pre_refr->pre_xlat_ysize = tiff.height;


	long long int warpdataSize = com.warpNum*com.warpDataLen;
	//if(count == 0)
	//pre_refr->pre_warp.resize(warpdataSize);
	//count ++;

	pre_refr->pre_warp_size = com.warpNum;
	pre_refr->pre_warp_data_size  = com.warpDataLen;
	pre_refr->pre_warp_data_xsize  = com.warpSize;
	pre_refr->pre_warp_data_ysize  = com.warpSize;

    //thrust::device_vector<Complex> d_warp_tmp(warpdataSize);
    //Complex *point_fftRes = thrust::raw_pointer_cast(&d_warp_tmp[0]);

    //thrust::device_vector<int> index_d(com.beginIndex.begin(),com.beginIndex.end());
    //int *index_p = thrust::raw_pointer_cast(&index_d[0]);
    thrust::device_vector<int> para_d(3);
	para_d[0] = com.warpDataLen;
	para_d[1] = com.warpSize;
	para_d[2] = tiff.width;

    getwarpdata_k<<<com.warpNum,com.warpSize>>>(tiff.fdata,tiff.warpdata,para_d[2] ,com.raw_beginIndex_d,para_d[1],para_d[0]);
    getWarpData(tiff,com.raw_point_warptmp,com);

    thrust::transform(com.d_warp_tmp.begin(),com.d_warp_tmp.end(),pre_refr->pre_warp.begin(),pre_refr->pre_warp.begin(),complex_plus_functor());


	if(DEBUG)
		cout << "prepare reference image for registration ok!" << endl;	
}
